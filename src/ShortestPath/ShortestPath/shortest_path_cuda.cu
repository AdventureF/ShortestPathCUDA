#include "hip/hip_runtime.h"
#include "shortest_path_cuda.cuh"

#define MAX_GRID_DIM_SIZE 65535

#define MAX_BLOCK_THREAD_COUNT 512

#define MAX_BLOCK_DIM_SIZE 512

#define INF 99999

#define ERR(err) (printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__),\
                    exit(EXIT_FAILURE))

__device__
int getWeight(int* graphMatrix, int n, int v, int u)
{
    return graphMatrix[n*v + u];
}

__global__
void shortestPathRelaxationKernel(int* graphMatrix, int* distance,
                                    int* local_distance, int* mask, int* previous, int n)
{
    int v = threadIdx.x + blockDim.x*blockIdx.x;
    
    // if tid is in bound of the vector and
    // if tid is in the mask
    if (v < n && mask[v]){
        // remove this vertex from mask
        mask[v] = 0;
        int u = 0;
        for (u = 0; u < n; u++){
            // get weight of each other vertex
            int weight = graphMatrix[n*v + u];
            // if it is a neighbor
            if (weight > 0){
                // Local distance relaxation
                if (local_distance[u] > distance[v] + weight){
                    local_distance[u] = distance[v] + weight;
                    previous[u] = v;
                }
                    
            }
        }
    }
}

__global__
void shortestPathUpdateDistanceKernel(int* graphMatrix, int* distance,
                                        int* local_distance, int* mask, int* previous, int n)
{
    int v = threadIdx.x + blockDim.x*blockIdx.x;
    if (v < n){
        if (distance[v] > local_distance[v]){
            distance[v] = local_distance[v];
            mask[v] = 1;
        }
        local_distance[v] = distance[v];
    }
}

/**
 * Returns 1 (true) if array is empty - all vertices have been marked 0,
 * 0 (false) otherwise
 */
__host__ 
int isEmpty(int* mask, int n)
{
    int i = 0;
    for (i = 0; i < n; i++){
        if (mask[i] == 1)
            return 0;
    }
    return 1;
}

__host__ 
void shortestPathLogic(int* d_graphMatrix, int* d_distance,
                       int* d_local_distance, int* d_mask, int* d_previous, int n)
{
    int size = n*sizeof(int);
    int sizeGraph= n*sizeof(int);

    int* mask = (int*)malloc(size);
    int* local_distance = (int*)malloc(size);
    int* graphMatrix = (int*)malloc(sizeGraph);

    hipError_t err;
    if ((err = hipMemcpy(mask, d_mask, size, hipMemcpyDeviceToHost)) != hipSuccess) ERR(err);
    if ((err = hipMemcpy(graphMatrix, d_graphMatrix, sizeGraph, hipMemcpyDeviceToHost)) != hipSuccess) ERR(err);

    int i = 0;

    for (i = 0; i<n; ++i)
    {
        int j = 0;
        printf("| ");
        for (j = 0; j<n; ++j)
        {
            printf("%d", graphMatrix[i*n + j]);
            printf(" ");
        }
        printf("|\n");
    }

    // TODO check if gridX is greater than MAX_GRID_DIM_SIZE
    int gridX = ((n - 1) / MAX_BLOCK_DIM_SIZE) + 1;
    int blockX = MAX_BLOCK_DIM_SIZE;

    dim3 DimGrid(gridX, 1, 1);
    dim3 DimBlock(blockX, 1, 1);

    printf("DimGrid: x: %d, y: %d, z: %d \n", DimGrid.x, DimGrid.y, DimGrid.z);
    printf("DimBlock: x: %d, y: %d, z: %d \n", DimBlock.x, DimBlock.y, DimBlock.z);

    while (isEmpty(mask, n) == 0){

        shortestPathRelaxationKernel << <DimGrid, DimBlock >> >(d_graphMatrix, d_distance, 
                                                                d_local_distance, d_mask, d_previous, n);

        if ((err = hipMemcpy(mask, d_mask, size, hipMemcpyDeviceToHost)) != hipSuccess) ERR(err);
        if ((err = hipMemcpy(local_distance, d_local_distance, size, hipMemcpyDeviceToHost)) != hipSuccess) ERR(err);

        printf("\nBefore Distance update \n\n");

        for (i = 0; i < n; i++){
            printf("mask[%d] = %d\n", i, mask[i]);
        }
        printf("\n");
        for (i = 0; i < n; i++){
            printf("local_distance[%d] = %d\n", i, local_distance[i]);
        }

        shortestPathUpdateDistanceKernel << <DimGrid, DimBlock >> >(d_graphMatrix, d_distance,
                                                                d_local_distance, d_mask, d_previous, n);

        if ((err = hipMemcpy(mask, d_mask, size, hipMemcpyDeviceToHost)) != hipSuccess) ERR(err);
        if ((err = hipMemcpy(local_distance, d_local_distance, size, hipMemcpyDeviceToHost)) != hipSuccess) ERR(err);

        printf("After Distance update \n\n");
        for (i = 0; i < n; i++){
            printf("mask[%d] = %d\n", i, mask[i]);
        }
        printf("\n");
        for (i = 0; i < n; i++){
            printf("local_distance[%d] = %d\n", i, local_distance[i]);
        }
    }
}

__host__
void shortestPathInit(int* graphMatrix, int n, int source, int* distance, int* previous)
{
    /********** INITIATE CPU ARRAYS **********/
    int graphSize = n*n*sizeof(int);
    int size = n*sizeof(int);
    // distance vector
    int* local_distance = (int*)malloc(size);
    // flag vector
    int* mask = (int*)malloc(size);

    int i = 0;
    for (i = 0; i < n; i++)
    {
        distance[i] = INF;
        local_distance[i] = INF;
        mask[i] = 0;
        previous[i] = -1;
    }

    printf("\n\n");
    for (i = 0; i<n; ++i)
    {
        int j = 0;
        printf("| ");
        for (j = 0; j<n; ++j)
        {
            printf("%d", graphMatrix[i*n + j]);
            printf(" ");
        }
        printf("|\n");
    }
    printf("\n\n");

    // mark source as visited
    mask[source] = 1;
    distance[source] = 0;
    local_distance[source] = 0;

    /*********** INITIATE DEVICE ARRAYS ***********/
    int* d_graphMatrix;
    int* d_distance;
    int* d_local_distance;
    int* d_mask;
    int* d_previous;

    hipError_t err;

    /*********** MEMORY ALLOCATION ***********/
    if ((err = hipMalloc((void**)&d_graphMatrix, graphSize)) != hipSuccess) ERR(err);
    if ((err = hipMalloc((void**)&d_distance, size)) != hipSuccess) ERR(err);
    if ((err = hipMalloc((void**)&d_local_distance, size)) != hipSuccess) ERR(err);
    if ((err = hipMalloc((void**)&d_mask, size)) != hipSuccess) ERR(err);
    if ((err = hipMalloc((void**)&d_previous, size)) != hipSuccess) ERR(err);

    /*********** COPY MEMORY ***********/
    if ((err = hipMemcpy(d_graphMatrix, graphMatrix, graphSize, hipMemcpyHostToDevice)) != hipSuccess) ERR(err);
    if ((err = hipMemcpy(d_distance, distance, size, hipMemcpyHostToDevice)) != hipSuccess) ERR(err);
    if ((err = hipMemcpy(d_local_distance, local_distance, size, hipMemcpyHostToDevice)) != hipSuccess) ERR(err);
    if ((err = hipMemcpy(d_mask, mask, size, hipMemcpyHostToDevice)) != hipSuccess) ERR(err);
    if ((err = hipMemcpy(d_previous, previous, size, hipMemcpyHostToDevice)) != hipSuccess) ERR(err);

    // work ...
    shortestPathLogic(d_graphMatrix, d_distance, d_local_distance, d_mask, d_previous, n);

    if ((err = hipMemcpy(distance, d_distance, size, hipMemcpyDeviceToHost)) != hipSuccess) ERR(err);
    if ((err = hipMemcpy(previous, d_previous, size, hipMemcpyDeviceToHost)) != hipSuccess) ERR(err);

    /*********** FREE MEMORY ***********/
    if ((err = hipFree(d_graphMatrix)) != hipSuccess) ERR(err);
    if ((err = hipFree(d_distance)) != hipSuccess) ERR(err);
    if ((err = hipFree(d_local_distance)) != hipSuccess) ERR(err);
    if ((err = hipFree(d_mask)) != hipSuccess) ERR(err);
    if ((err = hipFree(d_previous)) != hipSuccess) ERR(err);
}